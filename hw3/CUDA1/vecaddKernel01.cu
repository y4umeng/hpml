
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d\n", index);
    // Loop with stride of N
    for (int i = index; i < N * N; i += N) {
        C[i] = A[i] + B[i];
    }
}

