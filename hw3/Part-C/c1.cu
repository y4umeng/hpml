#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define H 1024
#define W 1024
#define C 3
#define FH 3
#define FW 3
#define K 64
#define P 1

// Kernel for performing convolution
__global__ void convolutionKernel(const double* __restrict__ I0, const double* __restrict__ F, double* O) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < W && y < H && k < K) {
        double sum = 0.0;
        for (int c = 0; c < C; ++c) {
            for (int i = 0; i < FH; ++i) {
                for (int j = 0; j < FW; ++j) {
                    int ix = x + i;
                    int iy = y + j;
                    sum += F[k * C * FH * FW + c * FH * FW + (FW - 1 - i) * FW + (FH - 1 - j)] * I0[c * (W + 2 * P) * (H + 2 * P) + iy * (W + 2 * P) + ix];
                }
            }
        }
        O[k * W * H + y * W + x] = sum;
    }
}

// Host function to initialize tensors
void initializeInput(double* I) {
    for (int c = 0; c < C; ++c) {
        for (int x = 0; x < H; ++x) {
            for (int y = 0; y < W; ++y) {
                I[c * H * W + x * W + y] = c * (x + y);
            }
        }
    }
}

void initializeFilter(double* F) {
    for (int k = 0; k < K; ++k) {
        for (int c = 0; c < C; ++c) {
            for (int i = 0; i < FH; ++i) {
                for (int j = 0; j < FW; ++j) {
                    F[k * C * FH * FW + c * FH * FW + i * FW + j] = (c + k) * (i + j);
                }
            }
        }
    }
}

void addPadding(const double* I, double* I0) {
    for (int c = 0; c < C; ++c) {
        for (int x = 0; x < H + 2 * P; ++x) {
            for (int y = 0; y < W + 2 * P; ++y) {
                if (x < P || x >= H + P || y < P || y >= W + P) {
                    I0[c * (H + 2 * P) * (W + 2 * P) + x * (W + 2 * P) + y] = 0.0;
                } else {
                    I0[c * (H + 2 * P) * (W + 2 * P) + x * (W + 2 * P) + y] = I[c * H * W + (x - P) * W + (y - P)];
                }
            }
        }
    }
}

int main() {
    // Allocate Unified Memory for input, filter, and output tensors
    double *I, *I0, *F, *O;
    hipMallocManaged(&I, C * H * W * sizeof(double));
    hipMallocManaged(&I0, C * (H + 2 * P) * (W + 2 * P) * sizeof(double));
    hipMallocManaged(&F, K * C * FH * FW * sizeof(double));
    hipMallocManaged(&O, K * W * H * sizeof(double));

    // Initialize input tensor I and filter F
    initializeInput(I);
    initializeFilter(F);
    addPadding(I, I0);

    // Define the block and grid sizes
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((W + blockDim.x - 1) / blockDim.x,
                 (H + blockDim.y - 1) / blockDim.y,
                 (K + blockDim.z - 1) / blockDim.z);

    // Run the convolution kernel and measure execution time
    auto start = std::chrono::high_resolution_clock::now();
    convolutionKernel<<<gridDim, blockDim>>>(I0, F, O);
    hipDeviceSynchronize();  // Ensure all threads are done
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate and display elapsed time
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Kernel execution time: " << elapsed.count() << " seconds" << std::endl;

    // Calculate checksum
    double checksum = 0.0;
    for (int i = 0; i < K * W * H; ++i) {
        checksum += O[i];
    }
    std::cout << "Checksum (sum of all elements in O): " << checksum << std::endl;

    // Free Unified Memory
    hipFree(I);
    hipFree(I0);
    hipFree(F);
    hipFree(O);

    return 0;
}
