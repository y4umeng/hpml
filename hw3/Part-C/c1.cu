#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define H 1024
#define W 1024
#define C 3
#define K 64
#define FH 3
#define FW 3
#define P 1

__global__ void tiledConvolutionKernel(const double* I0, const double* F, double* O, int width, int height) {
    extern __shared__ double tile[];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z;

    int threadX = threadIdx.x;
    int threadY = threadIdx.y;

    if (x < width && y < height) {
        double result = 0.0;
        
        for (int c = 0; c < C; ++c) {
            // Load input tile with padding into shared memory
            int input_x = x - P;
            int input_y = y - P;
            if (input_x >= 0 && input_x < width + 2 * P && input_y >= 0 && input_y < height + 2 * P) {
                tile[threadY * blockDim.x + threadX] = I0[c * (width + 2 * P) * (height + 2 * P) + input_y * (width + 2 * P) + input_x];
            } else {
                tile[threadY * blockDim.x + threadX] = 0.0;
            }

            __syncthreads();

            // Perform convolution
            for (int i = 0; i < FH; ++i) {
                for (int j = 0; j < FW; ++j) {
                    int tx = FW - 1 - i;
                    int ty = FH - 1 - j;
                    int ix = threadX + i;
                    int iy = threadY + j;

                    if (ix < blockDim.x && iy < blockDim.y) {
                        result += F[(k * C + c) * FH * FW + tx * FW + ty] * tile[iy * blockDim.x + ix];
                    }
                }
            }

            __syncthreads();
        }
        
        O[k * width * height + y * width + x] = result;
    }
}

void initializeInput(double* I) {
    for (int c = 0; c < C; ++c) {
        for (int x = 0; x < H; ++x) {
            for (int y = 0; y < W; ++y) {
                I[c * H * W + x * W + y] = c * (x + y);
            }
        }
    }
}

void initializeFilter(double* F) {
    for (int k = 0; k < K; ++k) {
        for (int c = 0; c < C; ++c) {
            for (int i = 0; i < FH; ++i) {
                for (int j = 0; j < FW; ++j) {
                    F[k * C * FH * FW + c * FH * FW + i * FW + j] = (c + k) * (i + j);
                }
            }
        }
    }
}

void addPadding(const double* I, double* I0) {
    for (int c = 0; c < C; ++c) {
        for (int x = 0; x < H + 2 * P; ++x) {
            for (int y = 0; y < W + 2 * P; ++y) {
                if (x < P || x >= H + P || y < P || y >= W + P) {
                    I0[c * (H + 2 * P) * (W + 2 * P) + x * (W + 2 * P) + y] = 0.0;
                } else {
                    I0[c * (H + 2 * P) * (W + 2 * P) + x * (W + 2 * P) + y] = I[c * H * W + (x - P) * W + (y - P)];
                }
            }
        }
    }
}

int main() {
    size_t imageSize = C * H * W * sizeof(double);
    size_t paddedSize = C * (H + 2 * P) * (W + 2 * P) * sizeof(double);
    size_t filterSize = K * C * FH * FW * sizeof(double);
    size_t outputSize = K * H * W * sizeof(double);

    double *I, *I0, *F, *O;
    hipMallocManaged(&I, imageSize);
    hipMallocManaged(&I0, paddedSize);
    hipMallocManaged(&F, filterSize);
    hipMallocManaged(&O, outputSize);

    initializeInput(I);
    initializeFilter(F);
    addPadding(I, I0);

    dim3 blockSize(16, 16);
    dim3 gridSize((W + blockSize.x - 1) / blockSize.x, (H + blockSize.y - 1) / blockSize.y, K);
    size_t sharedMemSize = blockSize.x * blockSize.y * sizeof(double);

    auto start = std::chrono::high_resolution_clock::now();
    tiledConvolutionKernel<<<gridSize, blockSize, sharedMemSize>>>(I0, F, O, W, H);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Kernel execution time: " << elapsed.count() << " seconds" << std::endl;

    double checksum = 0.0;
    for (int i = 0; i < K * W * H; ++i) {
        checksum += O[i];
    }
    std::cout << "Checksum (sum of all elements in O): " << checksum << std::endl;

    hipFree(I);
    hipFree(I0);
    hipFree(F);
    hipFree(O);

    return 0;
}
