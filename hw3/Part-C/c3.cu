#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdio.h>
#include <stdlib.h>

// Error checking macro
#define CHECK_CUDNN(call) do { \
    hipdnnStatus_t status = call; \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        printf("cuDNN error: %s\n", hipdnnGetErrorString(status)); \
        exit(1); \
    } \
} while(0)

#define CHECK_CUDA(call) do { \
    hipError_t status = call; \
    if (status != hipSuccess) { \
        printf("CUDA error: %s\n", hipGetErrorString(status)); \
        exit(1); \
    } \
} while(0)

// Problem dimensions
#define H 1024
#define W 1024
#define C 3
#define FH 3
#define FW 3
#define K 64
#define P 1

int main() {
    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    CHECK_CUDNN(hipdnnCreate(&cudnn));

    // Allocate host memory
    double *h_I = (double*)malloc(C * H * W * sizeof(double));
    double *h_F = (double*)malloc(K * C * FH * FW * sizeof(double));
    double *h_O = (double*)malloc(K * H * W * sizeof(double));
    
    // Initialize input tensor I
    for (int c = 0; c < C; c++) {
        for (int x = 0; x < H; x++) {
            for (int y = 0; y < W; y++) {
                h_I[c*H*W + x*W + y] = c * (x + y);
            }
        }
    }
    
    // Initialize filters F
    for (int k = 0; k < K; k++) {
        for (int c = 0; c < C; c++) {
            for (int i = 0; i < FH; i++) {
                for (int j = 0; j < FW; j++) {
                    h_F[k*C*FH*FW + c*FH*FW + i*FW + j] = (c + k) * (i + j);
                }
            }
        }
    }

    // Allocate device memory
    double *d_I, *d_F, *d_O;
    CHECK_CUDA(hipMalloc(&d_I, C * H * W * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_F, K * C * FH * FW * sizeof(double)));
    CHECK_CUDA(hipMalloc(&d_O, K * H * W * sizeof(double)));

    // Copy input data to device
    CHECK_CUDA(hipMemcpy(d_I, h_I, C * H * W * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_F, h_F, K * C * FH * FW * sizeof(double), hipMemcpyHostToDevice));

    // Create tensor descriptors
    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnTensorDescriptor_t output_descriptor;
    hipdnnFilterDescriptor_t filter_descriptor;
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));

    // Set tensor descriptors
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        input_descriptor,
        HIPDNN_TENSOR_NCHW,    // Format
        HIPDNN_DATA_DOUBLE,    // Data type
        1,                    // Batch size
        C,                    // Channels
        H,                    // Height
        W                     // Width
    ));

    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(
        filter_descriptor,
        HIPDNN_DATA_DOUBLE,    // Data type
        HIPDNN_TENSOR_NCHW,    // Format
        K,                    // Number of output feature maps
        C,                    // Number of input feature maps
        FH,                   // Filter height
        FW                    // Filter width
    ));

    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(
        convolution_descriptor,
        P, P,                 // Zero-padding height and width
        1, 1,                 // Vertical and horizontal stride
        1, 1,                 // Vertical and horizontal dilation
        HIPDNN_CONVOLUTION,    // Mode
        HIPDNN_DATA_DOUBLE    // Compute type
    ));

    // Get output dimensions
    int out_n, out_c, out_h, out_w;
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(
        convolution_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w
    ));

    // Set output descriptor
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        output_descriptor,
        HIPDNN_TENSOR_NCHW,    // Format
        HIPDNN_DATA_DOUBLE,    // Data type
        1,                    // Batch size
        K,                    // Channels
        out_h,               // Height
        out_w                // Width
    ));

    // Choose the best algorithm
    hipdnnConvolutionFwdAlgo_t algorithm;
    CHECK_CUDNN(hipdnnGetConvolutionForwardAlgorithm(
        cudnn,
        input_descriptor,
        filter_descriptor,
        convolution_descriptor,
        output_descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        0,  // No memory limit
        &algorithm
    ));

    // Get workspace size and allocate
    size_t workspace_size;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn,
        input_descriptor,
        filter_descriptor,
        convolution_descriptor,
        output_descriptor,
        algorithm,
        &workspace_size
    ));

    void* d_workspace;
    CHECK_CUDA(hipMalloc(&d_workspace, workspace_size));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Perform convolution
    const double alpha = 1.0;
    const double beta = 0.0;

    // Record start time
    CHECK_CUDA(hipEventRecord(start));

    CHECK_CUDNN(hipdnnConvolutionForward(
        cudnn,
        &alpha,
        input_descriptor, d_I,
        filter_descriptor, d_F,
        convolution_descriptor,
        algorithm,
        d_workspace, workspace_size,
        &beta,
        output_descriptor, d_O
    ));

    // Record stop time
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    // Calculate elapsed time
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Kernel execution time: %f ms\n", milliseconds);

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(h_O, d_O, K * H * W * sizeof(double), hipMemcpyDeviceToHost));

    // Calculate checksum
    double checksum = 0.0;
    for (int k = 0; k < K; k++) {
        for (int x = 0; x < H; x++) {
            for (int y = 0; y < W; y++) {
                checksum += h_O[k*H*W + x*W + y];
            }
        }
    }
    printf("Checksum: %.6e\n", checksum);

    // Cleanup
    CHECK_CUDA(hipFree(d_workspace));
    CHECK_CUDA(hipFree(d_I));
    CHECK_CUDA(hipFree(d_F));
    CHECK_CUDA(hipFree(d_O));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(input_descriptor));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(output_descriptor));
    CHECK_CUDNN(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(convolution_descriptor));
    CHECK_CUDNN(hipdnnDestroy(cudnn));
    
    free(h_I);
    free(h_F);
    free(h_O);

    return 0;
}