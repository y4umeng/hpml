#include <iostream>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <chrono>

#define H 1024
#define W 1024
#define C 3
#define K 64
#define FH 3
#define FW 3
#define P 1

void initializeInput(double* I) {
    for (int c = 0; c < C; ++c) {
        for (int x = 0; x < H; ++x) {
            for (int y = 0; y < W; ++y) {
                I[c * H * W + x * W + y] = c * (x + y);
            }
        }
    }
}

void initializeFilter(double* F) {
    for (int k = 0; k < K; ++k) {
        for (int c = 0; c < C; ++c) {
            for (int i = 0; i < FH; ++i) {
                for (int j = 0; j < FW; ++j) {
                    F[k * C * FH * FW + c * FH * FW + i * FW + j] = (c + k) * (i + j);
                }
            }
        }
    }
}

int main() {
    size_t imageSize = C * H * W * sizeof(double);
    size_t filterSize = K * C * FH * FW * sizeof(double);
    size_t outputSize = K * H * W * sizeof(double);

    double *I, *F, *O;
    hipMallocManaged(&I, imageSize);
    hipMallocManaged(&F, filterSize);
    hipMallocManaged(&O, outputSize);

    initializeInput(I);
    initializeFilter(F);

    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);

    cudnnTensorDescriptor_t inputDesc, outputDesc;
    cudnnFilterDescriptor_t filterDesc;
    cudnnConvolutionDescriptor_t convDesc;

    cudnnCreateTensorDescriptor(&inputDesc);
    cudnnCreateTensorDescriptor(&outputDesc);
    cudnnCreateFilterDescriptor(&filterDesc);
    cudnnCreateConvolutionDescriptor(&convDesc);

    cudnnSetTensor4dDescriptor(inputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_DOUBLE, 1, C, H, W);
    cudnnSetFilter4dDescriptor(filterDesc, CUDNN_DATA_DOUBLE, CUDNN_TENSOR_NCHW, K, C, FH, FW);
    cudnnSetConvolution2dDescriptor(convDesc, P, P, 1, 1, 1, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_DOUBLE);

    int outN, outC, outH, outW;
    cudnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc, &outN, &outC, &outH, &outW);
    cudnnSetTensor4dDescriptor(outputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_DOUBLE, outN, outC, outH, outW);

    int requestedAlgoCount = 1;
    cudnnConvolutionFwdAlgoPerf_t algoPerf;
    cudnnFindConvolutionForwardAlgorithm(cudnn, inputDesc, filterDesc, convDesc, outputDesc, requestedAlgoCount, &requestedAlgoCount, &algoPerf);
    
    size_t workspaceSize;
    cudnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc, convDesc, outputDesc, algoPerf.algo, &workspaceSize);

    void* workspace;
    hipMalloc(&workspace, workspaceSize);

    double alpha = 1.0, beta = 0.0;

    auto start = std::chrono::high_resolution_clock::now();
    cudnnConvolutionForward(cudnn, &alpha, inputDesc, I, filterDesc, F, convDesc, algoPerf.algo, workspace, workspaceSize, &beta, outputDesc, O);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    double checksum = 0.0;
    for (int i = 0; i < K * W * H; ++i) {
        checksum += O[i];
    }
    std::cout << "Checksum (sum of all elements in O): " << checksum << std::endl;

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Kernel execution time: " << elapsed.count() << " seconds" << std::endl;

    hipFree(I);
    hipFree(F);
    hipFree(O);
    hipFree(workspace);

    cudnnDestroyTensorDescriptor(inputDesc);
    cudnnDestroyTensorDescriptor(outputDesc);
    cudnnDestroyFilterDescriptor(filterDesc);
    cudnnDestroyConvolutionDescriptor(convDesc);
    cudnnDestroy(cudnn);

    return 0;
}
