#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void addKernel(const float* A, const float* B, float* C, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

void addVectorsOnGPU(float* h_A, float* h_B, float* h_C, size_t N, int numThreads) {
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate memory on the GPU
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    int numBlocks = (N + numThreads - 1) / numThreads;

    // Launch the kernel and measure time
    auto start = std::chrono::high_resolution_clock::now();
    addKernel<<<numBlocks, numThreads>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize(); // Wait for GPU to finish
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate and display elapsed time
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Time with " << numBlocks << " blocks and " << numThreads << " threads per block: "
              << elapsed.count() << " seconds" << std::endl;

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <K>" << std::endl;
        return 1;
    }

    int K = std::atoi(argv[1]);
    if (K <= 0) {
        std::cerr << "K must be a positive integer." << std::endl;
        return 1;
    }

    // Total number of elements
    size_t N = K * 1000000;

    // Allocate memory on the host
    float* h_A = (float*)malloc(N * sizeof(float));
    float* h_B = (float*)malloc(N * sizeof(float));
    float* h_C = (float*)malloc(N * sizeof(float));

    // Initialize arrays
    for (size_t i = 0; i < N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    std::cout << "Running with K = " << K << " million elements (" << N << " total elements)\n";

    // Scenario 1: One block with one thread
    std::cout << "\nScenario 1: One block with 1 thread" << std::endl;
    addVectorsOnGPU(h_A, h_B, h_C, N, 1);

    // Scenario 2: One block with 256 threads
    std::cout << "\nScenario 2: One block with 256 threads" << std::endl;
    addVectorsOnGPU(h_A, h_B, h_C, N, 256);

    // Scenario 3: Multiple blocks with 256 threads per block
    std::cout << "\nScenario 3: Multiple blocks with 256 threads per block" << std::endl;
    addVectorsOnGPU(h_A, h_B, h_C, N, 256);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}